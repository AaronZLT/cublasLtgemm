#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <cstdint>

#include "cublasLt_Ltgemm_fp8.h"
#include "helpers.h"

int roundoff(int v, int d)
{
    return (v + d - 1) / d * d;
}

/// Use hipblasLtMatmul to perform tensor-op Igemm with memory order transforms on all buffers
///
/// For better performance data order transforms should be offline as much as possible.
///
/// transa, transb assumed N; alpha, beta are host pointers, tensor ops allowed, alpha assumed 1, beta assumed 0,
/// stream assumed 0
void LtgemmTensor_fp8(hipblasLtHandle_t ltHandle,
                   int m,
                   int n,
                   int k,
                   const float *alpha, /* host pointer */
                   const __hip_fp8_e4m3_fnuz *A,
                   int lda,
                   const __hip_fp8_e4m3_fnuz *B,
                   int ldb,
                   const float *beta, /* host pointer */
                   float *C,
                   int ldc,
                   void *workspace,
                   size_t workspaceSize)
{
    INITTIMER
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    // float alpha = 1.0f, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    __hip_fp8_e4m3_fnuz *Atransform = NULL, *Btransform = NULL;
    float *Ctransform = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32 = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;
    hipblasLtOrder_t order_COL32_2R_4R4 = CUBLASLT_ORDER_COL32_2R_4R4;
    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void **>(&Atransform), sizeof(__hip_fp8_e4m3_fnuz) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void **>(&Btransform), sizeof(__hip_fp8_e4m3_fnuz) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void **>(&Ctransform), sizeof(float) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_8F_E4M3, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_8F_E4M3, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32_2R_4R4, sizeof(order_COL32_2R_4R4)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32F, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    /*NOT SUPPORT*/
    /*LTZHANG-TODO: Uncomment when SUPPORT*/
    // checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    /*NOT SUPPORT*/
    /*LTZHANG-TODO: Uncomment when SUPPORT*/
    //checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    START
    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     /*workspace,
                                     workspaceSize,
                                     ->
                                     NULL,
                                     NULL,*/
                                     workspace,
                                     workspaceSize,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));
    END_wo_print;
    printf("FP8-TensorCore M N K %d %d %d: %f ms\n", m, n, k, milliseconds);

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc)
        checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc)
        checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform)
        checkCudaStatus(hipFree(Ctransform));
    if (Btransform)
        checkCudaStatus(hipFree(Btransform));
    if (Atransform)
        checkCudaStatus(hipFree(Atransform));
}