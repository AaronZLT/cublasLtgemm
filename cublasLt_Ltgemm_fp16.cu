#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "cublasLt_Ltgemm_fp16.h"
#include "helpers.h"

/// Use hipblasLtMatmul to perform tensor-op Cgemm using planar complex memory layout and half-precision inputs.
///
/// For better performance data order transforms should be offline as much as possible.
///
/// transa, transb assumed N; alpha, beta are host pointers, tensor ops allowed, alpha assumed 1, beta assumed 0,
/// stream assumed 0
/// outputs can be either single or half precision, half precision is used in this example
void Ltgemm_fp16(hipblasLtHandle_t ltHandle,
                 int m,
                 int n,
                 int k,
                 const half *A,
                 int lda,
                 const half *B,
                 int ldb,
                 half *C,
                 int ldc)
{
    INITTIMER
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    half alpha = __float2half(1.0f);
    half beta = __float2half(0.0f);

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_16F, HIP_R_16F));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for planar complex matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, m, k, lda));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, k, n, ldb));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // Launch computation
    START
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     &beta,
                                     C,
                                     Cdesc,
                                     C,
                                     Cdesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));
    END_wo_print;
    printf("FP16-TensorCore M N K %d %d %d: %f ms\n", m, n, k, milliseconds);
    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc)
        checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc)
        checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
}